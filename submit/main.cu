#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <math.h>


using namespace std;

//*******************************************

// Write down the kernels here

__device__ __constant__ int gxcoord[1000];
__device__ __constant__ int gycoord[1000];

__global__ void finding_opponents(int *gRound, int T, int *ghp, int *gOpponents){
    if (*gRound % T == 0){
        return;
    }
    int attacker =  blockIdx.x;
    int k = threadIdx.x;
    int x4, y4,my,mx;
    __shared__ int opponent;
    __shared__ int nearest;
    __shared__ int x1, y1, x2, y2, slopex, slopey, x3, y3;

    if (threadIdx.x == 0) {
        opponent = -1;
        nearest = -1;
    }
    __syncthreads();
    if (threadIdx.x == 0 && ghp[attacker] > 0){
        opponent = (attacker + *gRound) % T;

        x1 = gxcoord[attacker];
        y1 = gycoord[attacker];
        x2 = gxcoord[opponent];
        y2 = gycoord[opponent];
        slopey = (y2 - y1);
        slopex = (x2 - x1);

        if ((x2 >= x1 && y2 >= y1)) {
            atomicExch(&x3,INT_MAX);
            atomicExch(&y3,INT_MAX);
        }
        else if ((x2 <= x1 && y2 >= y1)){
            atomicExch(&x3,INT_MIN);
            atomicExch(&y3,INT_MAX);
        }
        else if ((x2 <= x1 && y2 <= y1)) {
            atomicExch(&x3,INT_MIN);
            atomicExch(&y3,INT_MIN);
        }
        else if ((x2 >= x1 && y2 <= y1)){
            atomicExch(&x3,INT_MAX);
            atomicExch(&y3,INT_MIN);
        }
    }

    __syncthreads();
    if (ghp[attacker] > 0){
        
        if (k != attacker && ghp[k] > 0){
            x4 = gxcoord[k];
            y4 = gycoord[k];
            
            my = (y4 - y1);
            mx = (x4 - x1);
            
            if (slopey * mx == slopex * my){
                if ((x2 >= x1 && y2 >= y1) && (x4 >= x1 && y4 >= y1)){
                    if (x1 != x4) atomicMin(&x3, x4);
                    else atomicMin(&y3, y4);
                }
                else if ((x2 <= x1 && y2 >= y1) && (x4 <= x1 && y4 >= y1)){
                    if (x1 != x4) atomicMax(&x3, x4);
                    else atomicMin(&y3, y4);
                }
                else if ((x2 <= x1 && y2 <= y1) && (x4 <= x1 && y4 <= y1)){
                    if (x1 != x4) atomicMax(&x3, x4);
                    else atomicMax(&y3, y4);
                }
                else if ((x2 >= x1 && y2 <= y1) && (x4 >= x1 && y4 <= y1)){
                    if (x1 != x4) atomicMin(&x3, x4);
                    else atomicMax(&y3, y4);
                }
            }
        }
    }
    __syncthreads();
    if (ghp[attacker] > 0){
        if (x1 != x4){
            if (x3 == x4 && slopey * mx == slopex * my) nearest = k;
        }
        else {
            if (y3 == y4 && slopey * mx == slopex * my) nearest = k;
        }
    }



    __syncthreads();
    
    if (threadIdx.x == 0 && ghp[attacker] > 0){
        if (nearest == -1 || ghp[nearest] <= 0) gOpponents[attacker] = -1;
        else gOpponents[attacker] = nearest;
    }
}

__global__ void eval_score(int *gRound, int *ghp, int *gOpponents, int *gscore, int *gcount, int T){
    
    if (threadIdx.x == 0){
        (*gRound)++;
    }
    int attacker = threadIdx.x;
    if (ghp[attacker] <= 0) return;
    __syncthreads();
    int opponent = gOpponents[attacker];
    if ((*gRound - 1) % T != 0 && opponent != -1){
        if(atomicAdd(&ghp[opponent], -1) == 1) atomicAdd(gcount, 1);
        gscore[attacker]++;
    }
    
}

__global__ void setHP(int *ghp, int H){
    int id = threadIdx.x;
    ghp[id] = H;
}


//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;

    FILE *inputfilepointer;
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    if ( inputfilepointer == NULL )  {
        printf("input.txt file failed to open.");
        return 0; 
    }
    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************
    int *count = (int *) malloc(sizeof(int));
    *count = 0;

    int *gxcoord_cpy, *gycoord_cpy, *gscore, *ghp, *gOpponents, *gcount, *gRound;
    hipMalloc(&gxcoord_cpy, T * sizeof(int));
    hipMalloc(&gycoord_cpy, T * sizeof(int));
    hipMalloc(&gscore, T * sizeof(int));
    hipMemset(gscore, 0, T * sizeof(int));
    hipMalloc(&ghp, T * sizeof(int));
    hipMalloc(&gOpponents, T * sizeof(int));
    hipMalloc(&gcount, sizeof(int));
    hipMemset(gcount, 0, sizeof(int));
    hipMalloc(&gRound, sizeof(int));
    hipMemset(gRound, 0, sizeof(int));

    hipMemcpy(gxcoord_cpy, xcoord, T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gycoord_cpy, ycoord, T * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(gxcoord), gxcoord_cpy, T * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(gycoord), gycoord_cpy, T * sizeof(int));

    setHP<<<1, T>>>(ghp, H);

    while(*count + 1 != T && *count != T){
        
        finding_opponents<<<T, T>>>(gRound, T, ghp, gOpponents);
        eval_score<<<1, T>>>(gRound, ghp, gOpponents, gscore, gcount, T);
        hipMemcpy(count, gcount, sizeof(int), hipMemcpyDeviceToHost);
   
    }
    hipMemcpy(score, gscore, T * sizeof(int), hipMemcpyDeviceToHost);


    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}